#include "minisat/core/cuda.cuh"
#include "minisat/core/Solver.h"

void checkCudaError(const char msg[]) {
#ifdef CHECK_CUDA_ERROR
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        printf("Error message: %s\n", msg);
        exit(1);
    }
#endif
}

void Minisat::Solver::verifyUnsat(CRef cr) {
    if (cr == CREF_UNDEF) {
        if (cpuCheckConflict()) {
            printf("Miss unsat.\n");
            exit(1);
        }
    } else {
        Clause& c = ca[cr];
        for (int i = 0; i < c.size(); i++) {
            if (value(c[i]) != l_False) {
                printf("False unsat.\n");
                exit(1);
            }
        }
    }
}

bool Minisat::Solver::cpuCheckConflict() {
    for (int i = 0; i < clauses.size(); i++) {
        Clause& c = ca[clauses[i]];
        bool unsat = true;
        for (int j = 0; j < c.size(); j++) {
            if (value(c[j]) != l_False) {
                unsat = false;
                break;
            }
        }
        if (unsat) {
            return true;
        }
    }
    return false;
}
