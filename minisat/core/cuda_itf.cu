#include "minisat/core/cuda.cuh"
#include "minisat/core/Solver.h"
#include <vector>

using namespace Minisat;

void Solver::hostVecInit() {
#ifdef USE_CUDA
    hostClauseVec.clear();
    hostClauseEnd.clear();
    for (int i = 0; i < clauses.size(); i++) {
        CRef cr = clauses[i];
        Clause& c = ca[cr];
        if (c.learnt()) continue;
        for (int j = 0; j < c.size(); j++) {
            hostClauseVec.push_back(c[j]);
        }
        hostClauseEnd.push_back(hostClauseVec.size());
    }
#endif
}


void Solver::cudaClauseInit() {
#ifdef USE_CUDA
    size_t litCount = hostClauseVec.size();
    size_t clauseCount = hostClauseEnd.size();
    deviceClauseVec.init((unsigned*)hostClauseVec.data(), hostClauseVec.size());
    deviceClauseEnd.init((unsigned*)hostClauseEnd.data(), hostClauseEnd.size());
    deviceCRefs.init((unsigned*)clauses.data, clauses.size());
    checkCudaError("Failed to initialize memory for clause data.\n");
    hipMalloc(&deviceConfl, sizeof(unsigned));
    hipMalloc(&deviceAssigns, sizeof(uint8_t) * assigns.size());
    hipMalloc(&deviceImplCount, sizeof(unsigned));
    hipMalloc(&deviceImplications, sizeof(int) * assigns.size());
    hipMalloc(&deviceImplSource, sizeof(unsigned) * assigns.size());
    hipMalloc(&deviceLocks, sizeof(int) * assigns.size());
    checkCudaError("Failed to allocate memory for assgnment data.\n");
    hostImplications = (Lit*)malloc(sizeof(int) * assigns.size());
    hostImplSource = (CRef*)malloc(sizeof(unsigned) * assigns.size());
#endif
}

void Solver::cudaClauseFree() {
#ifdef USE_CUDA
    hipFree(deviceConfl);
    hipFree(deviceAssigns);
    hipFree(deviceImplCount);
    hipFree(deviceImplications);
    hipFree(deviceImplSource);
    hipFree(deviceLocks);
    checkCudaError("Failed to free device memory.\n");
    free(hostImplications);
    free(hostImplSource);
#endif
}

void Solver::cudaClauseUpdate() {
#ifdef USE_CUDA
    static unsigned updateCount = 0;
    updateCount++;
    unsigned originalClauseCount = clauses.size();
    unsigned originalLitCount = hostClauseVec.size();
    // Update CRefs of original clauses
    hipMemcpy(deviceCRefs.data, clauses.data, originalClauseCount * sizeof(unsigned), hipMemcpyHostToDevice);
    // Update learnt clauses
    cudaLearntUpdate();
    checkCudaError("Failed to update.\n");
#endif
}

void Solver::cudaLearntUpdate() {
    unsigned originalClauseCount = clauses.size();
    unsigned originalLitCount = hostClauseVec.size();
    // Clear learnt clauses
    hostLearntVec.clear();
    hostLearntEnd.clear();
    deviceClauseVec.resize(originalLitCount);
    deviceClauseEnd.resize(originalClauseCount);
    deviceCRefs.resize(originalClauseCount);
    // Add learnt clauses to host vector
    for (int i = 0; i < learnts.size(); i++) {
        CRef cr = learnts[i];
        Clause& c = ca[cr];
        for (int j = 0; j < c.size(); j++) {
            hostLearntVec.push_back(c[j]);
        }
        hostLearntEnd.push_back(originalLitCount + hostLearntVec.size());
    }
    // Copy host vector to device vector
    deviceClauseVec.bulk_push((unsigned*)hostLearntVec.data(), hostLearntVec.size());
    deviceClauseEnd.bulk_push((unsigned*)hostLearntEnd.data(), learnts.size());
    deviceCRefs.bulk_push((unsigned*)learnts.data, learnts.size());
}

void Solver::cudaAssignmentUpdate() {
#ifdef USE_CUDA
    hipMemcpy(deviceAssigns, assigns.begin(), sizeof(uint8_t) * assigns.size(), hipMemcpyHostToDevice);
    hipMemset(deviceLocks, 0, sizeof(int) * assigns.size());
#endif
}
