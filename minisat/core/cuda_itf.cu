#include "minisat/core/cuda.cuh"
#include "minisat/core/Solver.h"
#include <vector>

using namespace Minisat;

void Solver::hostVecInit() {
#ifdef USE_CUDA
    hostClauseVec.clear();
    hostClauseEnd.clear();
    for (int i = 0; i < clauses.size(); i++) {
        CRef cr = clauses[i];
        Clause& c = ca[cr];
        if (c.learnt()) continue;
        for (int j = 0; j < c.size(); j++) {
            hostClauseVec.push_back(c[j]);
        }
        c.sendToGPU();
        hostClauseEnd.push_back(hostClauseVec.size());
    }
#endif
}


void Solver::cudaClauseInit() {
#ifdef USE_CUDA
    size_t varCount = assigns.size();
    deviceClauseVec.init((unsigned*)hostClauseVec.data(), hostClauseVec.size());
    deviceClauseEnd.init((unsigned*)hostClauseEnd.data(), hostClauseEnd.size());
    deviceCRefs.init((unsigned*)clauses.data, clauses.size());
    checkCudaError("Failed to initialize memory for clause data.\n");
    hipMalloc(&deviceConflCount, sizeof(unsigned));
    hipMalloc(&deviceConfls, sizeof(unsigned) * MAX_CONFL);
    hipMalloc(&deviceAssigns, sizeof(uint8_t) * varCount);
    hipMalloc(&deviceImplCount, sizeof(unsigned));
    hipMalloc(&deviceImplications, sizeof(int) * varCount);
    hipMalloc(&deviceImplSource, sizeof(unsigned) * varCount);
    hipMalloc(&deviceLocks, sizeof(int) * varCount);
    checkCudaError("Failed to allocate memory for assgnment data.\n");
    hostImplications = (Lit*)malloc(sizeof(int) * varCount);
    hostImplSource = (CRef*)malloc(sizeof(unsigned) * varCount);
#endif
}

void Solver::cudaClauseFree() {
#ifdef USE_CUDA
    hipFree(deviceConflCount);
    hipFree(deviceConfls);
    hipFree(deviceAssigns);
    hipFree(deviceImplCount);
    hipFree(deviceImplications);
    hipFree(deviceImplSource);
    hipFree(deviceLocks);
    checkCudaError("Failed to free device memory.\n");
    free(hostImplications);
    free(hostImplSource);
#endif
}

void Solver::cudaClauseUpdate() {
#ifdef USE_CUDA
    static unsigned updateCount = 0;
    updateCount++;
    unsigned originalClauseCount = clauses.size();
    unsigned originalLitCount = hostClauseVec.size();
    // Update CRefs of original clauses
    hipMemcpy(deviceCRefs.data, clauses.data, originalClauseCount * sizeof(unsigned), hipMemcpyHostToDevice);
    // Update learnt clauses
    cudaLearntUpdate();
    checkCudaError("Failed to update.\n");
#endif
}

void Solver::cudaLearntUpdate() {
    unsigned originalClauseCount = clauses.size();
    unsigned originalLitCount = hostClauseVec.size();
    // Clear learnt clauses
    hostLearntVec.clear();
    hostLearntEnd.clear();
    deviceClauseVec.resize(originalLitCount);
    deviceClauseEnd.resize(originalClauseCount);
    deviceCRefs.resize(originalClauseCount);
    // Add learnt clauses to host vector
    for (int i = 0; i < learnts.size(); i++) {
        CRef cr = learnts[i];
        Clause& c = ca[cr];
        for (int j = 0; j < c.size(); j++) {
            hostLearntVec.push_back(c[j]);
        }
        hostLearntEnd.push_back(originalLitCount + hostLearntVec.size());
        c.sendToGPU();
    }
    // Copy host vector to device vector
    deviceClauseVec.bulk_push((unsigned*)hostLearntVec.data(), hostLearntVec.size());
    deviceClauseEnd.bulk_push((unsigned*)hostLearntEnd.data(), learnts.size());
    deviceCRefs.bulk_push((unsigned*)learnts.data, learnts.size());
}

void Solver::cudaLearntAppend(std::vector<CRef>& newLearnts) {
    hostLearntVec.clear();
    hostLearntEnd.clear();
    unsigned originalLitCount = deviceClauseVec.size;
    for (unsigned i = 0; i < newLearnts.size(); i++) {
        CRef cr = newLearnts[i];
        if (cr == CRef_Undef) continue;
        Clause& c = ca[cr];
        for (int j = 0; j < c.size(); j++) {
            hostLearntVec.push_back(c[j]);
        }
        hostLearntEnd.push_back(originalLitCount + hostLearntVec.size());
        c.sendToGPU();
    }
    deviceClauseVec.bulk_push((unsigned*)hostLearntVec.data(), hostLearntVec.size());
    deviceClauseEnd.bulk_push((unsigned*)hostLearntEnd.data(), hostLearntEnd.size());
    deviceCRefs.bulk_push((unsigned*)newLearnts.data(), newLearnts.size());
}

void Solver::cudaAssignmentUpdate() {
#ifdef USE_CUDA
    hipMemcpy(deviceAssigns, assigns.begin(), sizeof(uint8_t) * assigns.size(), hipMemcpyHostToDevice);
    hipMemset(deviceLocks, 0, sizeof(int) * assigns.size());
#endif
}
