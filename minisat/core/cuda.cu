#include "hip/hip_runtime.h"
#include "minisat/core/cuda.cuh"
#include "minisat/core/Solver.h"
#include <vector>

// interface (CPU) functions
using namespace Minisat;

/*_________________________________________________________________________________________________
|
|  propagate : [void]  ->  [Clause*]
|  
|  Description:
|    Propagates all enqueued facts. If a conflict arises, the conflicting clause is returned,
|    otherwise CRef_Undef.
|  
|    Post-conditions:
|      * the propagation queue is empty, even if there was a conflict.
|________________________________________________________________________________________________@*/
void Solver::propagate(std::vector<CRef>& hostConflicts) {
    CRef    confl     = CREF_UNDEF;
    int     num_props = 0;

    confl = checkConflictCaller(num_props);

    propagations += num_props;
    simpDB_props -= num_props;

    hostConflicts.clear();
    if (confl != CREF_UNDEF) hostConflicts.push_back(confl);
}
CRef Solver::propagate() {
    CRef    confl     = CRef_Undef;
    int     num_props = 0;

    while (qhead < trail.size()){
        Lit            p   = trail[qhead++];     // 'p' is enqueued fact to propagate.
        vec<Watcher>&  ws  = watches.lookup(p);
        int        i, j, end;
        num_props++;

        // for (i = 0; i < ws.size(); i++) {
        //     CRef     cr        = ws[i].cref;
        //     Clause& c = ca[cr];

        //     unsigned startIdx = 0;
        //     unsigned endIdx = c.size();
        //     bool unsat = true;
        //     for (j = startIdx; j < endIdx; j++) {
        //         Lit variable = c[j];
        //         if (value(variable) != l_False) {
        //             unsat = false;
        //             break;
        //         }
        //     }
        //     if (unsat) {
        //         confl = cr;
        //         break;
        //     }
        // }
        // if (confl != CRef_Undef) {
        //     i = j;
        //     break;
        // }
        for (i = j = 0, end = ws.size(); i < end; i++){
            // Try to avoid inspecting the clause:
            Lit blocker = ws[i].blocker;
            if (value(blocker) == l_True){
                ws[j++] = ws[i]; continue; }

            // Make sure the false literal is data[1]:
            CRef     cr        = ws[i].cref;
            Clause&  c         = ca[cr];
            Lit      false_lit = ~p;
            if (c[0] == false_lit) {
                c[0] = c[1], c[1] = false_lit;
            }
            assert(c[1] == false_lit);

            // If 0th watch is true, then clause is already satisfied.
            Lit     first = c[0];
            Watcher w     = Watcher(cr, first);
            assert(value(first) == VALUE(first.x, assigns.begin()));
            if (value(first) == l_True){
                ws[j++] = w; continue; }

            // Look for new watch:
            bool flag = false;
            for (int k = 2; k < c.size(); k++) {
                if (value(c[k]) != l_False){
                    c[1] = c[k]; c[k] = false_lit;
                    watches[~c[1]].push(w);
                    flag = true;
                    break;
                }
            }
            if (flag) continue;

            // Did not find watch -- clause is unit under assignment:
            ws[j++] = w;
            if (value(first) == l_False){
                confl = cr;
                qhead = trail.size();
                // Copy the remaining watches:
                i++;
                while (i < end)
                    ws[j++] = ws[i++];
                break;
            }else
                uncheckedEnqueue(first, cr);
        }
        ws.shrink(i - j);
    }
    propagations += num_props;
    simpDB_props -= num_props;

    return confl;
}

CRef Solver::checkConflictCaller(int& num_props) {
    
    CRef confl = CREF_UNDEF;
    unsigned implCount;
    while (true) {
        hipMemset(deviceConfl, 0xFF, sizeof(unsigned));
        hipMemset(deviceImplCount, 0, sizeof(unsigned));
        cudaAssignmentUpdate();
        checkCudaError("Failed to copy assignment data.\n");

        const size_t blockSize = 32;
        size_t gridSize = (clauses.size() - 1) / blockSize + 1;
        checkConflict<<<gridSize, blockSize>>>(
            (int*)deviceClauseVec.data, deviceClauseEnd.data, deviceCRefs.data,
            deviceCRefs.size, deviceAssigns, deviceLocks,
            deviceConfl, deviceImplications, deviceImplSource, deviceImplCount
        );
        checkCudaError("Error while launching kernel.\n");
        
        hipMemcpy(&confl, deviceConfl, sizeof(unsigned), hipMemcpyDeviceToHost);
        hipMemcpy(&implCount, deviceImplCount, sizeof(unsigned), hipMemcpyDeviceToHost);
        checkCudaError("Failed to copy data back.\n");
        hipDeviceSynchronize();
        // getUnitClauses();

        if (implCount > 0) {
            num_props += implCount;
            // Update variable assignment on the host side
            hipMemcpy(hostImplications, deviceImplications, sizeof(int) * implCount, hipMemcpyDeviceToHost);
            hipMemcpy(hostImplSource, deviceImplSource, sizeof(unsigned) * implCount, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            // printf("Found unit clause: ");
            // for (unsigned i = 0; i < implCount; i++) {
            //     printf(" %d ", hostImplSource[i]);
            // }
            // printf("\n");
            for (unsigned i = 0; i < implCount; i++) {
                CRef cr = hostImplSource[i];
                Clause& c = ca[cr];
                uncheckedEnqueue(hostImplications[i], hostImplSource[i]);
                for (int k = 0; k < c.size(); k++) {
                    if (value(c[k]) == l_True) {
                        Lit tmp = c[k];
                        c[k] = c[0];
                        c[0] = tmp;
                        break;
                    }
                }
            }
        }
        if (implCount == 0 || confl != CREF_UNDEF) break;
    }

    return confl;
}
// Cuda device functions

__global__ void checkConflict(int* clauses, unsigned* ends, unsigned* crefs, unsigned clauseCount,
    uint8_t* assigns, int* lock, unsigned* conflict, int* implications, unsigned* implSource, unsigned* implCount) {
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= clauseCount) return;

    unsigned startIdx = (idx == 0) ? 0 : ends[idx-1];
    unsigned endIdx = ends[idx];
    unsigned valCount[4];
    int implied = LIT_UNDEF;
    for (unsigned i = 0; i < 4; i++) {
        valCount[i] = 0;
    }
    for (unsigned i = startIdx; i < endIdx; i++) {
        uint8_t value = VALUE(clauses[i], assigns);
        valCount[value]++;
        if (value >= LU) implied = clauses[i];
    }
    if (valCount[LF] == endIdx - startIdx - 1 && valCount[LT] == 0) {
        // Found a unit clause
        if (atomicExch(lock+VAR(implied), 1) == 0) {
            // Obtain the lock and set the value
            // assigns[VAR(implied)] = SIGN(implied);
            unsigned writeIdx = atomicAdd(implCount, 1);
            implications[writeIdx] = implied;
            implSource[writeIdx] = crefs[idx];
        } else if (VALUE(implied, assigns) == LF) {
            // Failed to obtain lock.
            // conflict
            valCount[LF] = endIdx - startIdx;
        }
    }
    if (valCount[LF] == endIdx - startIdx) {
        // Fount a conflicting clause (evaluates to 0)
        unsigned cr = crefs[idx];
        atomicCAS(conflict, CREF_UNDEF, cr);
    }
}
