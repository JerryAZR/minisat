#include "hip/hip_runtime.h"
#include "minisat/core/cuda.cuh"
#include "minisat/core/Solver.h"
#include <vector>

// interface (CPU) functions
using namespace Minisat;

/*_________________________________________________________________________________________________
|
|  propagate : [void]  ->  [Clause*]
|  
|  Description:
|    Propagates all enqueued facts. If a conflict arises, the conflicting clause is returned,
|    otherwise CRef_Undef.
|  
|    Post-conditions:
|      * the propagation queue is empty, even if there was a conflict.
|________________________________________________________________________________________________@*/
#ifdef USE_CUDA
CRef Solver::propagate() {
    CRef    confl     = CREF_UNDEF;
    int     num_props = 0;

    while (qhead < trail.size()){
        Lit            p   = trail[qhead++];     // 'p' is enqueued fact to propagate.
        vec<Watcher>&  ws  = watches.lookup(p);
        int        i, j, end;
        num_props++;
        
        // First check for conflicts
        bool run_cuda = (hostClauseEnd.size() > 0) && (ws.size() > 32);
        if (run_cuda) {
            confl = checkConflictCaller();
            // testCheckConflict(
            //     (int*) hostClauseVec.data(), hostClauseEnd.data(),
            //     (unsigned*) clauses.data, (unsigned) clauses.size(),
            //     (uint8_t*) assigns.begin(), (unsigned*) (&confl));
        }
        
        if (confl == CREF_UNDEF) {
            confl = CRef_Undef;
            // std::vector<Lit> tmpLits;
            // std::vector<CRef> tmpCRefs;
            for (i = j = 0, end = ws.size(); i < end; i++){
                // Try to avoid inspecting the clause:
                Lit blocker = ws[i].blocker;
                if (value(blocker) == l_True){
                    ws[j++] = ws[i]; continue; }

                // Make sure the false literal is data[1]:
                CRef     cr        = ws[i].cref;
                Clause&  c         = ca[cr];
                Lit      false_lit = ~p;
                if (c[0] == false_lit) {
                    c[0] = c[1], c[1] = false_lit;
                }
                assert(c[1] == false_lit);

                // If 0th watch is true, then clause is already satisfied.
                Lit     first = c[0];
                Watcher w     = Watcher(cr, first);
                assert(value(first) == VALUE(first.x, assigns.begin()));
                if (value(first) == l_True){
                    ws[j++] = w; continue; }

                // Look for new watch:
                bool flag = false;
                for (int k = 2; k < c.size(); k++) {
                    if (value(c[k]) != l_False){
                        c[1] = c[k]; c[k] = false_lit;
                        watches[~c[1]].push(w);
                        flag = true;
                        break;
                    }
                }
                if (flag) continue;

                // Did not find watch -- clause is unit under assignment:
                ws[j++] = w;
                if (value(first) == l_False){
                    confl = cr;
                    qhead = trail.size();
                    // Copy the remaining watches:
                    i++;
                    while (i < end)
                        ws[j++] = ws[i++];
                    break;
                }else {
                    // tmpCRefs.push_back(cr);
                    // tmpLits.push_back(first);
                    uncheckedEnqueue(first, cr);
                }
            }
            // for (unsigned n = 0; n < tmpLits.size(); n++) {
            //     if (value(tmpLits[n]) == l_Undef)
            //         uncheckedEnqueue(tmpLits[n], tmpCRefs[n]);
            // }
        } else {
            qhead = trail.size();
            i = j = 0;
            break;
        }
        ws.shrink(i - j);
    }
    propagations += num_props;
    simpDB_props -= num_props;

    return confl;
}
#else
CRef Solver::propagate() {
    CRef    confl     = CRef_Undef;
    int     num_props = 0;

    while (qhead < trail.size()){
        Lit            p   = trail[qhead++];     // 'p' is enqueued fact to propagate.
        vec<Watcher>&  ws  = watches.lookup(p);
        int        i, j, end;
        num_props++;

        // for (i = 0; i < ws.size(); i++) {
        //     CRef     cr        = ws[i].cref;
        //     Clause& c = ca[cr];

        //     unsigned startIdx = 0;
        //     unsigned endIdx = c.size();
        //     bool unsat = true;
        //     for (j = startIdx; j < endIdx; j++) {
        //         Lit variable = c[j];
        //         if (value(variable) != l_False) {
        //             unsat = false;
        //             break;
        //         }
        //     }
        //     if (unsat) {
        //         confl = cr;
        //         break;
        //     }
        // }
        // if (confl != CRef_Undef) {
        //     i = j;
        //     break;
        // }
        for (i = j = 0, end = ws.size(); i < end; i++){
            // Try to avoid inspecting the clause:
            Lit blocker = ws[i].blocker;
            if (value(blocker) == l_True){
                ws[j++] = ws[i]; continue; }

            // Make sure the false literal is data[1]:
            CRef     cr        = ws[i].cref;
            Clause&  c         = ca[cr];
            Lit      false_lit = ~p;
            if (c[0] == false_lit) {
                c[0] = c[1], c[1] = false_lit;
            }
            assert(c[1] == false_lit);

            // If 0th watch is true, then clause is already satisfied.
            Lit     first = c[0];
            Watcher w     = Watcher(cr, first);
            assert(value(first) == VALUE(first.x, assigns.begin()));
            if (value(first) == l_True){
                ws[j++] = w; continue; }

            // Look for new watch:
            bool flag = false;
            for (int k = 2; k < c.size(); k++) {
                if (value(c[k]) != l_False){
                    c[1] = c[k]; c[k] = false_lit;
                    watches[~c[1]].push(w);
                    flag = true;
                    break;
                }
            }
            if (flag) continue;

            // Did not find watch -- clause is unit under assignment:
            ws[j++] = w;
            if (value(first) == l_False){
                confl = cr;
                qhead = trail.size();
                // Copy the remaining watches:
                i++;
                while (i < end)
                    ws[j++] = ws[i++];
                break;
            }else
                uncheckedEnqueue(first, cr);
        }
        ws.shrink(i - j);
    }
    propagations += num_props;
    simpDB_props -= num_props;

    return confl;
}
#endif

CRef Solver::checkConflictCaller() {
    CRef confl;
    unsigned implCount;
    hipMemset(deviceConfl, 0xFF, sizeof(unsigned));
    hipMemset(deviceImplCount, 0, sizeof(unsigned));
    cudaAssignmentUpdate();
    checkCudaError("Failed to copy assignment data.\n");

    const size_t blockSize = 32;
    size_t gridSize = (clauses.size() - 1) / blockSize + 1;
    checkConflict<<<gridSize, blockSize>>>(
        (int*)deviceClauseVec.data, deviceClauseEnd.data, deviceCRefs.data,
        deviceCRefs.size, deviceAssigns, deviceLocks,
        deviceConfl, deviceImplications, deviceImplSource, deviceImplCount
    );
    checkCudaError("Error while launching kernel.\n");
    
    hipMemcpy(&confl, deviceConfl, sizeof(unsigned), hipMemcpyDeviceToHost);
    hipMemcpy(&implCount, deviceImplCount, sizeof(unsigned), hipMemcpyDeviceToHost);
    checkCudaError("Failed to copy data back.\n");
    hipDeviceSynchronize();

    if (implCount > 0) {
        // Update variable assignment on the host side
        hipMemcpy(hostImplications, deviceImplications, sizeof(uint8_t) * implCount, hipMemcpyDeviceToHost);
        hipMemcpy(hostImplSource, deviceImplSource, sizeof(unsigned) * implCount, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        for (unsigned i = 0; i < implCount; i++) {
            if (assigns.size() <= var(hostImplications[i])) {
                printf("Variable %d out of range %d\n", var(hostImplications[i]), assigns.size());
            }
            assert(value(hostImplications[i]) == l_Undef);
            uncheckedEnqueue(hostImplications[i], hostImplSource[i]);
        }
    }

    return confl;
}
// Cuda device functions

__global__ void checkConflict(int* clauses, unsigned* ends, unsigned* crefs, unsigned clauseCount,
    uint8_t* assigns, int* lock, unsigned* conflict, uint8_t* implications, unsigned* implSource, unsigned* implCount) {
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= clauseCount) return;

    unsigned startIdx = (idx == 0) ? 0 : ends[idx-1];
    unsigned endIdx = ends[idx];
    unsigned valCount[4];
    int implied = LIT_UNDEF;
    for (unsigned i = 0; i < 4; i++) {
        valCount[i] = 0;
    }
    for (unsigned i = startIdx; i < endIdx; i++) {
        uint8_t value = VALUE(clauses[i], assigns);
        valCount[value]++;
        if (value >= LU) implied = clauses[i];
    }
    if (valCount[LF] == endIdx - startIdx - 1 && valCount[LT] == 0) {
        // Found a unit clause
        if (atomicExch(lock+VAR(implied), 1) == 0) {
            // Obtain the lock and set the value
            assigns[VAR(implied)] = SIGN(implied) ^ 1;
            unsigned writeIdx = atomicAdd(implCount, 1);
            implications[writeIdx] = implied;
            implSource[writeIdx] = crefs[idx];
        } else if (VALUE(implied, assigns) == LF) {
            // Failed to obtain lock.
            // conflict
            valCount[LF] = endIdx - startIdx;
        }
    }
    if (valCount[LF] == endIdx - startIdx) {
        // Fount a conflicting clause (evaluates to 0)
        unsigned cr = crefs[idx];
        atomicCAS(conflict, CREF_UNDEF, cr);
    }
}
