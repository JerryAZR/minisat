#include "hip/hip_runtime.h"
#include "minisat/core/cuda.cuh"
#include "minisat/core/Solver.h"
#include <vector>
#include <stdlib.h>

using std::vector;
#define CUDATEST

// interface (CPU) functions
using namespace Minisat;

void propagateTest(int clauseCount, int* clauses, int* clausesEnd, uint8_t* assigns, int* actions);

/*_________________________________________________________________________________________________
|
|  propagate : [void]  ->  [Clause*]
|  
|  Description:
|    Propagates all enqueued facts. If a conflict arises, the conflicting clause is returned,
|    otherwise CRef_Undef.
|  
|    Post-conditions:
|      * the propagation queue is empty, even if there was a conflict.
|________________________________________________________________________________________________@*/
#ifdef CUDATEST
CRef Solver::propagate()
{
    CRef    confl     = CRef_Undef;
    int     num_props = 0;

    while (qhead < trail.size()){
        Lit            p   = trail[qhead++];     // 'p' is enqueued fact to propagate.
        vec<Watcher>&  ws  = watches.lookup(p);
        num_props++;

        int* hostEnds = (int*)malloc(ws.size() * sizeof(int));
        CRef* hostCrefs = (CRef*)malloc(ws.size() * sizeof(CRef));
        vector<int> hostClauses;
        int totalSize = 0;
        int clauseCount = 0;
        int watcherCount = 0;
        // Determine the size of each clause
        for (int idx = 0; idx < ws.size(); idx++) {
            Lit blocker = ws[idx].blocker;
            if (value(blocker) == l_True) {
                Watcher tmp = ws[watcherCount];
                ws[watcherCount++] = ws[idx];
                ws[idx] = tmp;
                continue;
            }
            Clause& cl = ca[ws[idx].cref];
            Lit false_lit = ~p;
            if (cl[0] == false_lit) {
                cl[0] = cl[1];
                cl[1] = false_lit;
            }
            totalSize += cl.size();
            hostCrefs[clauseCount] = ws[idx].cref;
            hostEnds[clauseCount++] = totalSize;
            for (int litIdx = 0; litIdx < cl.size(); litIdx++) {
                hostClauses.push_back(toInt(cl[litIdx]));
            }
        }
        int* hostActions = (int*)malloc(clauseCount * sizeof(int));
        if (clauseCount) {
            propagateTest(clauseCount, hostClauses.data(), hostEnds, (uint8_t*)assigns.begin(), hostActions);
        //     int* deviceActions, * deviceClauses, * deviceEnds;
        //     uint8_t* deviceAssigns;
        //     int* numConflicts, * conflictIndices;
        //     hipMalloc(&deviceActions, clauseCount * sizeof(int));
        //     hipMalloc(&deviceClauses, totalSize * sizeof(int));
        //     hipMalloc(&deviceEnds, clauseCount * sizeof(int));
        //     hipMalloc(&deviceAssigns, assigns.size() * sizeof(uint8_t));
        //     hipMalloc(&conflictIndices, clauseCount * sizeof(int));
        //     hipMalloc(&numConflicts, sizeof(int));
        //     hipMemset(numConflicts, 0, sizeof(int));

        //     hipMemcpy(deviceEnds, hostEnds, clauseCount * sizeof(int), hipMemcpyHostToDevice);
        //     hipMemcpy(deviceClauses, hostClauses.data(), totalSize * sizeof(int), hipMemcpyHostToDevice);
        //     hipMemcpy(deviceAssigns, assigns.begin(), assigns.size() * sizeof(uint8_t), hipMemcpyHostToDevice);

        //     const size_t blockSize = 32;
        //     size_t gridSize = ((totalSize-1) / blockSize) + 1;
        //     propagateKernel<<<gridSize, blockSize>>>(clauseCount, deviceClauses, deviceEnds, deviceAssigns, deviceActions);
        //     hipDeviceSynchronize();

        //     hipMemcpy(hostActions, deviceActions, clauseCount * sizeof(int), hipMemcpyDeviceToHost);
        //     hipMemcpy(hostClauses.data(), deviceClauses, totalSize * sizeof(int), hipMemcpyDeviceToHost);

        //     hipFree(deviceClauses);
        //     hipFree(deviceAssigns);
        //     hipFree(deviceEnds);
        //     hipFree(deviceActions);
        //     hipFree(numConflicts);
        //     hipFree(conflictIndices);
        }
        
        // Iterate over the returned clauses and update the data on CPU
        int i, j; // i: read index; j: write index.
        Watcher* wsBegin = &ws[watcherCount];
        for (i = j = 0; i < clauseCount; i++) {
            int action = hostActions[i];
            CRef cr = hostCrefs[i];
            int beginIdx = (i == 0) ? 0 : hostEnds[i-1];
            int* c = hostClauses.data() + beginIdx;
            Lit first;
            first.x = c[0];
            Watcher w = Watcher(cr, first);
            switch (action)
            {
            case CL_NEWBLOCK: wsBegin[j++] = w; break;
            case CL_NEWWATCH: 
                Lit second;
                second.x = c[1];
                watches[~second].push(w);
                assert(ca[cr].size() == (hostEnds[i] - beginIdx));
                memcpy(&((ca[cr])[0]), c, ca[cr].size() * sizeof(int));
                break;
            case CL_UNIT: 
                if (value(first) == l_Undef) {
                    uncheckedEnqueue(first, cr);
                }
                wsBegin[j++] = w;
                break;
            case CL_CONFLICT:
                wsBegin[j++] = w;
                i++;
                confl = cr;
                qhead = trail.size();
                // Copy the remaining watches:
                while (i < clauseCount) {
                    wsBegin[j++] = wsBegin[i++];                }
                break;
            default: break;
            }
        }
        free(hostEnds);
        free(hostCrefs);
        free(hostActions);
        ws.shrink(i - j);
    }
    propagations += num_props;
    simpDB_props -= num_props;

    return confl;
}
#else
CRef Solver::propagate()
{
    CRef    confl     = CRef_Undef;
    int     num_props = 0;

    while (qhead < trail.size()){
        Lit            p   = trail[qhead++];     // 'p' is enqueued fact to propagate.
        vec<Watcher>&  ws  = watches.lookup(p);
        Watcher        *i, *j, *end;
        num_props++;

        // Testing preprocessing
        int watcherCount = 0;
        for (int idx = 0; idx < ws.size(); idx++) {
            Lit blocker = ws[idx].blocker;
            if (value(blocker) == l_True) {
                Watcher tmp = ws[watcherCount];
                ws[watcherCount++] = ws[idx];
                ws[idx] = tmp;
                continue;
            }
            Clause& cl = ca[ws[idx].cref];
            Lit false_lit = ~p;
            if (cl[0] == false_lit) {
                cl[0] = cl[1];
                cl[1] = false_lit;
            }
        }

        vector<Lit> tmpAssignments;
        vector<CRef> tmpReason;
        assert(tmpAssignments.size() == 0);
        for (i = j = (Watcher*)ws, end = i + ws.size(); i != end; i++){
            // Try to avoid inspecting the clause:
            Lit blocker = i->blocker;
            if (value(blocker) == l_True){
                *j++ = *i; continue; }

            // Make sure the false literal is data[1]:
            CRef     cr        = i->cref;
            Clause&  c         = ca[cr];
            Lit      false_lit = ~p;
            if (c[0] == false_lit) {
                c[0] = c[1], c[1] = false_lit;
            }
            assert(c[1] == false_lit);

            // If 0th watch is true, then clause is already satisfied.
            Lit     first = c[0];
            Watcher w     = Watcher(cr, first);
            assert(value(first) == VALUE(first.x, assigns.begin()));
            if (value(first) == l_True){
                *j++ = w; continue; }

            // Look for new watch:
            bool flag = false;
            for (int k = 2; k < c.size(); k++) {
                if (value(c[k]) != l_False){
                    c[1] = c[k]; c[k] = false_lit;
                    watches[~c[1]].push(w);
                    flag = true;
                    break;
                }
            }
            if (flag) continue;

            // Did not find watch -- clause is unit under assignment:
            *j++ = w;
            if (value(first) == l_False){
                confl = cr;
                qhead = trail.size();
                // Copy the remaining watches:
                i++;
                while (i < end)
                    *j++ = *i++;
                break;
            }else {
                tmpAssignments.push_back(first);
                tmpReason.push_back(cr);
                // uncheckedEnqueue(first, cr);
            }
        }
        for (int idx = 0; idx < tmpAssignments.size(); idx++) {
            if (value(tmpAssignments[idx]) == l_Undef) {
                uncheckedEnqueue(tmpAssignments[idx], tmpReason[idx]);
            }
        }
        ws.shrink(i - j);
    }
    propagations += num_props;
    simpDB_props -= num_props;

    return confl;
}

#endif

// Cuda device functions
__global__ void propagateKernel(int clauseCount, int* clauses, int* clausesEnd, uint8_t* assigns, int* actions) {
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < clauseCount) {
        int clauseStart = (idx == 0) ? 0 : clausesEnd[idx - 1];
        int* c = clauses + clauseStart;
        int c_size = clausesEnd[idx] - clauseStart;
        int first = c[0];
        if (VALUE(first, assigns) == LT) {
            // update blcker
            actions[idx] = CL_NEWBLOCK;
            return;
        }
        // Look for new watch (i.e unassigned variable):
        for (int k = 2; k < c_size; k++) {
            if (VALUE(c[k], assigns) != LF) {
                int temp = c[1];
                c[1] = c[k];
                c[k] = temp;
                // watch new literal
                actions[idx] = CL_NEWWATCH;
                return;
            }
        }
        if (VALUE(first, assigns) == LF) {
            // conflict
            actions[idx] = CL_CONFLICT;
        } else {
            // unit clause;
            actions[idx] = CL_UNIT;
        }
    }
}

/* checkWatcher
 * Possible outcomes:
 * - Clause already satisfied (blocker == true) -- do nothing
 * - Clause already satisfied (blocker != true) -- update current blocker
 * - Clause undetermined (At least 2 unassigned vars) -- watch another literal
 * - Clause unit -- unit propagation
 * - Clause unsatisfied -- keep the remaining clauses and terminate
 * Return true if watcher needs to be updated. false if not
 */
int checkWatcher(int blocker, int p, int* c, unsigned c_size, uint8_t* assigns) {
    if (VALUE(blocker, assigns) == LT) {
        return CL_NOCHANGE; // Clause Satisfied
    }
    
    int false_lit = p ^ 1;
    if (c[0] == false_lit) {
        c[0] = c[1];
        c[1] = false_lit;
    }
    assert(c[1] == false_lit);

    int first = c[0];
    if (VALUE(first, assigns) == LT) {
        // update blcker
        return CL_NEWBLOCK;
    }

    // Look for new watch (i.e unassigned variable):
    for (unsigned k = 2; k < c_size; k++) {
        if (VALUE(c[k], assigns) != LF) {
            c[1] = c[k]; c[k] = false_lit;
            // watch new literal
            return CL_NEWWATCH;
        }
    }

    if (VALUE(first, assigns) == LF) {
        // conflict
        return CL_CONFLICT;
    } else {
        // unit clause;
        return CL_UNIT;
    }
}

void propagateTest(int clauseCount, int* clauses, int* clausesEnd, uint8_t* assigns, int* actions) {
    for(unsigned idx = 0; idx < clauseCount; idx++) {
        int clauseStart = (idx == 0) ? 0 : clausesEnd[idx - 1];
        int* c = clauses + clauseStart;
        int c_size = clausesEnd[idx] - clauseStart;
        int first = c[0];
        if (VALUE(first, assigns) == LT) {
            // update blcker
            actions[idx] = CL_NEWBLOCK;
            continue;
        }
        // Look for new watch (i.e unassigned variable):
        bool flag = false;
        for (int k = 2; k < c_size; k++) {
            if (VALUE(c[k], assigns) != LF) {
                int temp = c[1];
                c[1] = c[k];
                c[k] = temp;
                // watch new literal
                actions[idx] = CL_NEWWATCH;
                flag = true;
                break;
            }
        }
        if (flag) continue;
        if (VALUE(first, assigns) == LF) {
            // conflict
            actions[idx] = CL_CONFLICT;
        } else {
            // unit clause;
            actions[idx] = CL_UNIT;
        }
    }
}