#include "minisat/core/deviceVector.h"

using namespace Minisat;

void cudaDeviceMemcpy(void* dest, void* src, unsigned size);

deviceVector::deviceVector(unsigned size, unsigned cap) : data(nullptr), size(size), cap(cap) {
    if (cap) {
        hipMalloc(&data, cap * sizeof(unsigned));
    }
}

deviceVector::~deviceVector() {
    if (cap) {
        hipFree(data);
    }
}

// Initialize deviceVector with a given host array and size
void deviceVector::init(unsigned* hostData, unsigned sz) {
    if (cap < sz) {
        while (cap < sz) cap = cap << 1;
        if (data) hipFree(data);
        hipMalloc(&data, cap * sizeof(unsigned));
    }
    size = sz;
    hipMemcpy(data, hostData, size * sizeof(unsigned), hipMemcpyHostToDevice);
}

// Push an array of data
void deviceVector::bulk_push(unsigned* hostData, unsigned sz) {
    unsigned newSize = sz + size;
    reserve(newSize);
    hipMemcpy(data+size, hostData, sz * sizeof(unsigned), hipMemcpyHostToDevice);
    size = newSize;
}

// Avoid using this if possible
void deviceVector::push(unsigned newData) {
    reserve(size+1);
    hipMemcpy(data+size, &newData, sizeof(unsigned), hipMemcpyHostToDevice);
    size++;
}

// Resize the vector
void deviceVector::resize(unsigned newSize) {
    reserve(newSize);
    size = newSize;
}

// Raise cap to at least newCap
// May allocate more memory, but does not change the size
void deviceVector::reserve(unsigned newCap) {
    if (cap <= newCap) {
        // Calculate new cap
        while (cap <= newCap) cap = cap << 1;
        // Allocate more memory
        unsigned* tmpPtr = data;
        hipMalloc(&data, cap * sizeof(unsigned));
        // Copy existing data
        if (size) {
            cudaDeviceMemcpy(data, tmpPtr, size * sizeof(unsigned));
            hipFree(tmpPtr);
        }
    }
}

void cudaDeviceMemcpy(void* dest, void* src, unsigned size) {
    hipMemcpy(dest, src, size * sizeof(unsigned), hipMemcpyDeviceToDevice);
}
